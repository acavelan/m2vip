#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "utils.h"

#define CUCHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ unsigned char clamp(float value, int min, int max)
{
	if(value < min)
		return 0;
	else if(value > max)
		return max;
	else
		return value;
}

__global__ void RGBToYCBCR(const unsigned char *pixels, float *ycbcrimg, unsigned char *yimg, int width, int height)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x < width && y < height)
	{
		/*
		const float KB = 0.0593;
		const float KR = 0.2627;
		const float KG = 1 - KB - KR;
		
		unsigned char R = pixels[(y * width + x) * 3 + 0];
		unsigned char G = pixels[(y * width + x) * 3 + 1];
		unsigned char B = pixels[(y * width + x) * 3 + 2]; 
		
		float Y = KR * R + (1 - KR - KB) * G + KB * B;
		float CB = (B - Y) / (1 + KR + KG - KB);
		float CR = (R - Y) / (1 - KR + KG + KB);

		yimg[y * width + x] = clamp(Y, 0, 255);

		ycbcrimg[(y * width + x)*3 + 0] = Y;
		ycbcrimg[(y * width + x)*3 + 1] = CB;
		ycbcrimg[(y * width + x)*3 + 2] = CR;
		*/
		
		unsigned char R = pixels[(y * width + x) * 3 + 0];
		unsigned char G = pixels[(y * width + x) * 3 + 1];
		unsigned char B = pixels[(y * width + x) * 3 + 2]; 
		
		float Y = 0.2627f * R + 0.678f * G + 0.0593f * B;
		float CB = (B - Y) / 1.8814f;
		float CR = (R - Y) / 1.4746f;

		yimg[y * width + x] = clamp(Y, 0, 255);
		//ycbcrimg[(y * width + x)*3 + 0] = Y;
		ycbcrimg[(y * width + x)*3 + 1] = CB;
		ycbcrimg[(y * width + x)*3 + 2] = CR;
	}
}

__global__ void YCBCRToRGB(const float *ycbcrimg, unsigned char *yimg, float *lut, unsigned char *pixels, int width, int height)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x < width && y < height)
	{
		/*
		const float KB = 0.0593;
		const float KR = 0.2627;
		const float KG = 1 - KB - KR;
		
		float Y = lut[yimg[y * width + x]];
		float CB = ycbcrimg[(y * width + x) * 3 + 1];
		float CR = ycbcrimg[(y * width + x) * 3 + 2];
		
		unsigned char R = clamp((Y + CR * (1 - KR + KG + KB)), 0, 255);
		unsigned char B = clamp((Y + CB * (1 + KR + KG - KB)), 0, 255);
		unsigned char G = clamp((Y - KR * R - KB * B) / (1 - KB - KR), 0, 255);
		
		yimg[y * width + x] = Y;
		pixels[(y * width + x) * 3 + 0] = R;
		pixels[(y * width + x) * 3 + 1] = G;
		pixels[(y * width + x) * 3 + 2] = B;
		*/
		
		float Y = lut[yimg[y * width + x]];
		float CB = ycbcrimg[(y * width + x) * 3 + 1];
		float CR = ycbcrimg[(y * width + x) * 3 + 2];
		
		unsigned char R = clamp(Y + CR * 1.4746f, 0, 255);
		unsigned char B = clamp(Y + CB * 1.8814f, 0, 255);
		unsigned char G = clamp((Y - 0.2627f * R - 0.0593f * B) / 0.678, 0, 255);
		
		pixels[(y * width + x) * 3 + 0] = R;
		pixels[(y * width + x) * 3 + 1] = G;
		pixels[(y * width + x) * 3 + 2] = B;
	}
}

__global__ void histogram(unsigned char *yimg, unsigned int *yhist, int threads, int width, int height)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int size = (width * height) / threads;
	
	unsigned int *histBase = (yhist + tid * 256);
	
	if(tid < threads)
	{	
		int start = tid * size;
		int end = start + size;
		for(int i=start; i < end; ++i)
		{
			//histBase[yimg[i]]++;
			atomicAdd(&histBase[yimg[i]], 1);
		}
	}
}

__global__ void reduction(unsigned int *yhist, int threads, int width, int height, int level)
{
	unsigned int tid = (blockIdx.x * blockDim.x + threadIdx.x) * level;
	
	if(tid < threads && tid % (level * 2) == 0 && tid + level < threads)
	{		
		for(int i=0; i<256; i++)
		{
			(yhist + tid * 256)[i] += (yhist + (tid + level) * 256)[i];
		}
	}
}

int main(int argc, char **argv)
{
	struct timeval start, last, now, computation;
	
	gettimeofday(&start, 0);
	
	if(argc < 2)
	{
		printf("usage: %s image\n", argv[0]);
		return 0;
	}
	
	ilInit();
	ilEnable(IL_ORIGIN_SET);
	ilEnable(IL_FILE_OVERWRITE);
	
	ILboolean result = ilLoadImage(argv[1]);

	if(!result)
	{
		ILenum err = ilGetError() ;
		printf("Failed to load %s\n", argv[1]);
		printf("Error: %s\n", ilGetString(err));
	}

	ilConvertImage(IL_RGB, IL_UNSIGNED_BYTE);
	ilOriginFunc(IL_ORIGIN_UPPER_LEFT);
		
	ILuint width = ilGetInteger(IL_IMAGE_WIDTH);
	ILuint height = ilGetInteger(IL_IMAGE_HEIGHT);
	
	unsigned int size = width * height;
	
	ILubyte * pixels = ilGetData();
	
	gettimeofday(&now, 0);
	computation = last = now;
	
	printf("Image (%d * %d) loaded in %f\n", width, height, get_time(start, now));
	
	unsigned char *eqpixels = (unsigned char*)malloc(size * 3 * sizeof(unsigned char));
	
	printf("Size: %d * %d\n", width, height);
	
	///=================================================================
	int threads = 1024;
		
	unsigned char *d_pixels = 0, *d_eqpixels = 0;
	unsigned char *d_yimg = 0;
	unsigned int *d_yhist = 0;
	float *d_ycbcrimg = 0;
	float *d_lut = 0;
	
	CUCHECK(hipMalloc((void**)&d_pixels, size * 3 * sizeof(unsigned char)));
	CUCHECK(hipMalloc((void**)&d_ycbcrimg, size * 3 * sizeof(float)));
	CUCHECK(hipMalloc((void**)&d_yimg, size * sizeof(unsigned char)));
	CUCHECK(hipMalloc((void**)&d_yhist, threads * 256 * sizeof(unsigned int)));
	CUCHECK(hipMalloc((void**)&d_lut, 256 * sizeof(float)));
	
	hipMemcpy(d_pixels, pixels, size * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemset(d_yhist, 0, threads * 256 * sizeof(unsigned int));
	
	// RGB -> YCBCR
	dim3 blockDim(16, 16, 1);
	dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1);
	RGBToYCBCR<<<gridDim, blockDim, 0>>>(d_pixels, d_ycbcrimg, d_yimg, width, height);
	
	hipFree(d_pixels);
	
	hipDeviceSynchronize();
	gettimeofday(&now, 0);
	printf("\tRGB -> YCBCR %f\n", get_time(last, now));
	last = now;
	
	blockDim = dim3(8, 1, 1);
	gridDim = dim3((threads + blockDim.x - 1) / blockDim.x, 1, 1);
	histogram<<<gridDim, blockDim, 0>>>(d_yimg, d_yhist, threads, width, height);
	
	hipDeviceSynchronize();
	gettimeofday(&now, 0);
	printf("\tHistogram computed in %f\n", get_time(last, now));
	last = now;
	
	for(int level=1; level<=threads; level*=2)
	{
		gridDim = dim3((threads / level + blockDim.x - 1) / blockDim.x, 1, 1);
		reduction<<<gridDim, blockDim, 0>>>(d_yhist, threads, width, height, level);
	}
	
	hipDeviceSynchronize();
	gettimeofday(&now, 0);
	printf("\tReduction computed in %f\n", get_time(last, now));
	last = now;
	
	unsigned int *yhist = (unsigned int*)malloc(256 * threads * sizeof(unsigned int));
	
	hipMemcpy(yhist, d_yhist, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	// Equalization
	float ylut[256];
	double ysum = 0;
	for(unsigned int i = 0; i < 256; ++i)
	{
		ysum += (float)yhist[i] / size;
		ylut[i] = ysum * 255;
	}
	
	hipMemcpy(d_lut, ylut, 256 * sizeof(float), hipMemcpyHostToDevice);
	free(yhist);
	
	hipDeviceSynchronize();
	gettimeofday(&now, 0);
	printf("\tEqualization computed in %f\n", get_time(last, now));
	last = now;
	
	CUCHECK(hipMalloc((void**)&d_eqpixels, size * 3 * sizeof(unsigned char)));
	
	// YCBCR -> RGB
	blockDim = dim3(16, 16, 1);
	gridDim = dim3((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1);
	YCBCRToRGB<<<gridDim, blockDim, 0>>>(d_ycbcrimg, d_yimg, d_lut, d_eqpixels, width, height);

	hipMemcpy(eqpixels, d_eqpixels, size * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	
	hipFree(d_eqpixels);
	hipFree(d_ycbcrimg);
	hipFree(d_yimg);
	hipFree(d_yhist);
	hipFree(d_lut);
	
	hipDeviceSynchronize();
	gettimeofday(&now, 0);
	printf("\tYCBCR -> RGb in %f\n", get_time(last, now));
	printf("Equalization computed in %f\n", get_time(computation, now));
	last = now;
	
	///=================================================================
	
	// Save images
	save_image("equal.jpg", eqpixels, width, height);
	free(eqpixels);
	
	gettimeofday(&now, 0);
	printf("Result saved in %f\n", get_time(last, now));
	printf("Total time %f\n", get_time(start, now));

	return 0;
}
