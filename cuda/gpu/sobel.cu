#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "utils.h"

__global__ void apply_grayscale(unsigned char *grayimg, const unsigned char *image, int width, int height)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x < width && y < height)
	{
		const unsigned char R = image[(y * width + x)*3 + 0];
		const unsigned char G = image[(y * width + x)*3 + 1];
		const unsigned char B = image[(y * width + x)*3 + 2]; 

		unsigned char gray = (307 * R + 604 * G + 113 * B) >> 10;

		grayimg[y * width + x] = gray;
	}
}

__global__ void apply_sobel(const unsigned char *input, unsigned char *output, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x > 1 && y > 1 && y < height - 1 && x < width -1)
	{
		#define PX(kx) (kx + x)
		#define PY(ky) width * (ky + y)
		
		int gx = 0, gy = 0;
		
		gx += input[PX(-1) + PY(-1)];
		gx += 2 * input[PX(-1) + PY(0)];
		gx += input[PX(-1) + PY(1)];
		gx -= input[PX(1) + PY(-1)];
		gx -= 2 * input[PX(1) + PY(0)];
		gx -= input[PX(1) + PY(1)];
		
		gy += input[PX(-1) + PY(-1)];
		gy += 2 * input[PX(0) + PY(-1)];
		gy += input[PX(1) + PY(-1)];
		gy -= input[PX(-1) + PY(1)];
		gy -= 2 * input[PX(0) + PY(1)];
		gy -= input[PX(1) + PY(1)];
		
		float grad = sqrt(float(gx * gx + gy * gy));
		output[y * width + x] = grad > 255 ? 255 : grad;
	}
}
	
int main(int argc, char **argv)
{
	struct timeval start, last, now, computation;
	
	gettimeofday(&start, 0);
	
	if(argc < 2)
	{
		printf("usage: %s image\n", argv[0]);
		return 0;
	}
	
	ilInit();
	ilEnable(IL_ORIGIN_SET);
	ilEnable(IL_FILE_OVERWRITE);
	
	ILboolean result = ilLoadImage(argv[1]);

	if(!result)
	{
		ILenum err = ilGetError() ;
		printf("Failed to load %s\n", argv[1]);
		printf("Error: %s\n", ilGetString(err));
	}

	ilConvertImage(IL_RGB, IL_UNSIGNED_BYTE);
	ilOriginFunc(IL_ORIGIN_UPPER_LEFT);
		
	ILuint width = ilGetInteger(IL_IMAGE_WIDTH);
	ILuint height = ilGetInteger(IL_IMAGE_HEIGHT);
	
	unsigned int size = width * height;
	
	ILubyte *image = ilGetData();
	
	gettimeofday(&now, 0);
	computation = last = now;
	
	printf("Image (%d * %d) loaded in %f\n", width, height, get_time(start, now));
	
	unsigned char *d_image = 0, *d_grayimg = 0;
	unsigned char *grayimg = (unsigned char*)malloc(size * sizeof(unsigned char));
	
	hipMalloc((void**)&d_image, size * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_grayimg, size * sizeof(unsigned char));
    
	hipMemcpy(d_image, (unsigned char*)image, size * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
	
	// Execution du kernel
	dim3 blockDim(16, 16, 1);
	dim3 gridDim((width + blockDim.x - 1)/ blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1);
	
	apply_grayscale<<<gridDim, blockDim, 0>>>(d_grayimg, d_image, width, height);
	
	hipFree(d_image);
	
	gettimeofday(&now, 0);
	printf("Gray image computed in %f\n", get_time(last, now));
	last = now;
	
	unsigned char *d_sobel = 0;
	unsigned char *sobel = (unsigned char*)malloc(size * sizeof(unsigned char));
	
	hipMalloc((void**)&d_sobel, size * sizeof(unsigned char));
	
	apply_sobel<<<gridDim, blockDim, 0>>>(d_grayimg, d_sobel, width, height);
	
	hipMemcpy(sobel, d_sobel, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	
	hipFree(d_sobel);
	hipFree(d_grayimg);
	
	hipDeviceSynchronize();
	
	gettimeofday(&now, 0);
	printf("Sobel completed in %f\n", get_time(computation, now));
	last = now;
	
	hipDeviceReset();
	
	// Save images	
	save_gray_image("sobel.jpg", sobel, width, height);

	free(sobel);
	
	gettimeofday(&now, 0);
	printf("Result saved in %f\n", get_time(last, now));
	printf("Total time %f\n", get_time(start, now));

	return 0;
}
