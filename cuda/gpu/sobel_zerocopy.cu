#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "utils.h"

#define CUCHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	#ifdef _DEBUG_CUDA_
	   if (code != hipSuccess) 
	   {
		  fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		  if (abort) exit(code);
	   }
   #endif
}

__global__ void apply_grayscale(unsigned char *grayimg, const unsigned char *image, int width, int height)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x < width && y < height)
	{
		const unsigned char R = image[(y * width + x)*3 + 0];
		const unsigned char G = image[(y * width + x)*3 + 1];
		const unsigned char B = image[(y * width + x)*3 + 2]; 

		unsigned char gray = (307 * R + 604 * G + 113 * B) >> 10;

		grayimg[y * width + x] = gray;
	}
}

__global__ void apply_sobel(const unsigned char *input, unsigned char *output, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x > 1 && y > 1 && y < height - 1 && x < width -1)
	{
		#define PX(kx) (kx + x)
		#define PY(ky) width * (ky + y)
		
		int gx = 0, gy = 0;
		
		gx += input[PX(-1) + PY(-1)];
		gx += 2 * input[PX(-1) + PY(0)];
		gx += input[PX(-1) + PY(1)];
		gx -= input[PX(1) + PY(-1)];
		gx -= 2 * input[PX(1) + PY(0)];
		gx -= input[PX(1) + PY(1)];
		
		gy += input[PX(-1) + PY(-1)];
		gy += 2 * input[PX(0) + PY(-1)];
		gy += input[PX(1) + PY(-1)];
		gy -= input[PX(-1) + PY(1)];
		gy -= 2 * input[PX(0) + PY(1)];
		gy -= input[PX(1) + PY(1)];
		
		float grad = sqrt(float(gx * gx + gy * gy));
		output[y * width + x] = grad > 255 ? 255 : grad;
	}
}

int main(int argc, char **argv)
{
	struct timeval start, last, now, computation;
	
	gettimeofday(&start, 0);
	
	if(argc < 2)
	{
		printf("usage: %s image\n", argv[0]);
		return 0;
	}
	
	ilInit();
	ilEnable(IL_ORIGIN_SET);
	ilEnable(IL_FILE_OVERWRITE);

	ILboolean result = ilLoadImage(argv[1]);

	if(!result)
	{
		ILenum err = ilGetError() ;
		printf("Failed to load %s\n", argv[1]);
		printf("Error: %s\n", ilGetString(err));
	}

	ilConvertImage(IL_RGB, IL_UNSIGNED_BYTE);
	ilOriginFunc(IL_ORIGIN_UPPER_LEFT);
		
	ILuint width = ilGetInteger(IL_IMAGE_WIDTH);
	ILuint height = ilGetInteger(IL_IMAGE_HEIGHT);
	
	unsigned int size = width * height;
	
	ILubyte *ILimage = ilGetData();
	
	gettimeofday(&now, 0);
	computation = last = now;
	
	unsigned char *d_image = 0;
	
	hipHostMalloc((void**)&d_image, size * 3 * sizeof(unsigned char), hipHostMallocMapped);
	//hipHostMalloc((void**)&image, size * 3 * sizeof(unsigned char));
	memcpy(d_image, ILimage, size * 3 * sizeof(unsigned char));
	
	printf("Image (%d * %d) loaded in %f\n", width, height, get_time(start, now));
	
	const int nStreams = 4;
	int streamSize = (height / nStreams) * width;
	
	hipStream_t streams[nStreams];
	
	for(int i=0; i<nStreams; i++)
		CUCHECK(hipStreamCreate(&streams[i]));
		
	unsigned char *d_grayimg = 0;
	unsigned char *grayimg = (unsigned char*)malloc(size * sizeof(unsigned char));
	unsigned char *d_sobel = 0;
	unsigned char *sobel = (unsigned char*)malloc(size * sizeof(unsigned char));
	
	CUCHECK(hipMalloc((void**)&d_grayimg, size * sizeof(unsigned char)));
    CUCHECK(hipMalloc((void**)&d_sobel, size * sizeof(unsigned char)));
    
    // Execution du kernel
	dim3 blockDim(16, 16, 1);
	dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height / nStreams + blockDim.y - 1) / blockDim.y, 1);
	
	for (int i = 0; i < nStreams; ++i)
	{
		int offset = i * streamSize;
		apply_grayscale<<<gridDim, blockDim, 0, streams[i]>>>(&d_grayimg[offset], &d_image[offset * 3], width, height / nStreams);
	}
	
	for(int i=0; i<nStreams; i++)
		CUCHECK(hipStreamDestroy(streams[i]));
	
	gridDim = dim3((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1);
	apply_sobel<<<gridDim, blockDim, 0>>>(d_grayimg, d_sobel, width, height);
	
	CUCHECK(hipMemcpy(sobel, d_sobel, size * sizeof(unsigned char), hipMemcpyDeviceToHost));
	
	CUCHECK(hipFree(d_sobel));
	CUCHECK(hipFree(d_grayimg));
	CUCHECK(hipHostFree(d_image));
	
	hipDeviceSynchronize();
	
	gettimeofday(&now, 0);
	printf("Sobel completed in %f\n", get_time(computation, now));
	last = now;
	
	hipDeviceReset();
	
	// Save images	
	save_gray_image("sobel.jpg", sobel, width, height);

	free(sobel);
	
	gettimeofday(&now, 0);
	printf("Result saved in %f\n", get_time(last, now));
	printf("Total time %f\n", get_time(start, now));

	return 0;
}
